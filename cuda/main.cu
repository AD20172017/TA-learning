#include "hip/hip_runtime.h"
#include<iostream>
#include "time.h"
#include "vec3.h"
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__
void render(vec3 *fb, int max_x, int max_y){
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;
    if((i>max_x)||(j>=max_y)) return;
    int pixel_index = j*max_x + i;
    fb[pixel_index] = vec3( double (i) / max_x, double (j) / max_y, 0.2f);
}

int main() {

    // Image
    bool in= false;
    std::cerr<<"need write size of image?"<<std::endl;
    std::cin>>in;

    int image_width = 1600;
    int image_height = 1000;

    int block_x=8,block_y=8;

    if(in){
        std::cerr<<"image_width: ";
        std::cin>>image_width;
        std::cerr<<'\n';

        std::cerr<<"image_height: ";
        std::cin>>image_height;
        std::cerr<<'\n';
    }
    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << block_x << "x" << block_y << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    //统一内存
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    clock_t start, stop;
    start = clock();

    // Render
    dim3 blocks(image_width/block_x+1,image_height/block_y+1);
    dim3 threads(block_x,block_y);
    render<<<blocks, threads>>>(fb, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = image_height-1; j >= 0; j--) {
        std::clog << "\rScanlines remaining: ------------------" << j << ' ' << std::flush;
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j*image_width + i;
            int ir = int(255.99*fb[pixel_index].x());
            int ig = int(255.99*fb[pixel_index].y());
            int ib = int(255.99*fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    std::clog << "\nDone-------------------------------------\n";

    checkCudaErrors(hipFree(fb));
}